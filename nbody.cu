#include <hip/hip_runtime.h>

#include <iostream>

int main() {
  hipError_t err;
  int const N = 1<<20;

  float* x;
  err = hipMallocManaged(&x, N * sizeof(float));
  if (err != hipSuccess) {
    std::cout << "Cannot allocate x: " << err << "\n";
    return 1;
  }
}
